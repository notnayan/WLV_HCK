/*
Student Name: Nayan Raj Khanal
University ID: 2227486

This CUDA C program utilizes parallel processing on a GPU to crack an encrypted password. 
It employs a CUDA kernel, decryptPass, to generate potential password combinations and decrypt them using a function called Cudacrack. 
The code efficiently searches for the correct password by leveraging GPU threads. 
The main function initializes character sets, allocates memory on both the CPU and GPU, launches the kernel, and prints the results, displaying the original encrypted password and the decrypted password if found. 
The program demonstrates a parallelized approach to password decryption using CUDA.

1.	Generate encrypted password in the kernel function (using CudaCrypt function) to be compared to original encrypted password:
The Cudacrack function in the CUDA kernel is responsible for generating an encrypted password based on a given raw password. 
The transformation involves adding and subtracting specific values to each character in the raw password. 
The resulting encrypted password is then checked against the target encrypted password.

2.	Allocating the correct amount of memory on the GPU based on input data. Memory is freed once used:
Memory is allocated on the GPU for the alphabet, numbers, encrypted password, and the output password. 
The correct amount of memory is allocated based on the size of the data (alphabet, numbers, encrypted password, and output password). 
Memory is freed once it is no longer needed.

3.	Program works with multiple blocks and threads: 
The CUDA kernel decryptPass is designed to work with multiple blocks and threads. 
The kernel is launched with a 2D grid (dim3(26, 26, 1)) and 2D block (dim3(10, 10, 1)) configuration to explore different combinations of alphabet and numbers.

4.	Decrypted password sent back to the CPU and printed:
If a thread finds a matching password, it prints the result, including the encrypted and decrypted passwords. 
The decrypted password is sent back to the CPU, and the results are printed on the CPU.

*/

// Header files

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

//__global__ --> GPU function which can be launched by many blocks and threads
//__device__ --> GPU function or variables
//__host__ --> CPU function or variables

__device__ char* Cudacrack(char* rawPassword) {
    char * newPassword = (char *) malloc(sizeof(char) * 11);

    newPassword[0] = rawPassword[0] + 2;
    newPassword[1] = rawPassword[0] - 2;
    newPassword[2] = rawPassword[0] + 1;
    newPassword[3] = rawPassword[1] + 3;
    newPassword[4] = rawPassword[1] - 3;
    newPassword[5] = rawPassword[1] - 1;
    newPassword[6] = rawPassword[2] + 2;
    newPassword[7] = rawPassword[2] - 2;
    newPassword[8] = rawPassword[3] + 4;
    newPassword[9] = rawPassword[3] - 4;
    newPassword[10] = '\0';

    for(int i = 0; i < 10; i++){
        if(i >= 0 && i < 6){ // checking all lowercase letter limits
            if(newPassword[i] > 122){
                newPassword[i] = (newPassword[i] - 122) + 97;
            } else if(newPassword[i] < 97){
                newPassword[i] = (97 - newPassword[i]) + 97;
            }
        } else { // checking number section
            if(newPassword[i] > 57){
                newPassword[i] = (newPassword[i] - 57) + 48;
            } else if(newPassword[i] < 48){
                newPassword[i] = (48 - newPassword[i]) + 48;
            }
        }
    }
    return newPassword;
}

// CUDA kernel to check potential passwords against the encrypted password
__device__ int arePassEqual(char* one, char* two, int length) {
    int result = 1;
    for (int i = 0; i < length; i++) {
        if (one[i] != two[i]) {
            result = 0;
            break;
        }
    }
    return result;
}

// CUDA kernel to perform password decryption using Cudacrack on all possible combinations
__global__ void decryptPass(char* alphabet, char* numbers, char* encryptedPass, char* deviceOutputPass) {
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;

    // Check if a password has already been found by other threads
    if(*deviceOutputPass != '\0') {
        return;
    }

    // Create a potential password combination based on thread indices
    char potentialPass[4];
    potentialPass[0] = alphabet[blockIdx.x];
    potentialPass[1] = alphabet[blockIdx.y];
    potentialPass[2] = numbers[threadIdx.x];
    potentialPass[3] = numbers[threadIdx.y];

    // Apply Cudacrack transformations to the potential password
    char* encryptedPotential = Cudacrack(potentialPass);

    // Check if the encrypted potential password matches the target encrypted password
    if (arePassEqual(encryptedPass, encryptedPotential, 11)) {
        // Print the result if a match is found and copy the password to the output
        printf("ThreadID '%d' found password of '%s' is '%s'\n", threadID, encryptedPass, potentialPass);
        for (int i = 0; i < 4; i++ ) {
            deviceOutputPass[i] = potentialPass[i];
        }
    }

    free(encryptedPotential); // Free allocated memory
}

// Main function
int main(int argc, char** argv) {

	/*
		Test Encrypted Passwords:
		x y 3 3 = zvycvx5171
    p p 1 3 = rnqsmo3171
		s p 1 8 = uqtsmo3134
	*/
    // Encrypted password
    const char* encryptedPass = "rnqsmo3171"; //pp13
    if (argc > 1) {
        encryptedPass = argv[1];
    }
    printf("Encrypted Password: '%s'\n", encryptedPass);
    printf("---\n");

    // Character sets for alphabet and numbers
    char cpuAlphabet[26] = { 'a','b','c','d','e','f','g','h','i','j','k','l','m','n','o','p','q','r','s','t','u','v','w','x','y','z' };
    char cpuNumbers[10] = { '0', '1', '2', '3', '4', '5', '6' ,'7', '8', '9' };

    // Size of the encrypted password
    int sizeOfEncryptedPass = sizeof(char) * 11;

    // Allocate and copy memory for alphabet, numbers, encrypted password, and output password on GPU
    char* gpuAlphabet;
    hipMalloc((void**) &gpuAlphabet, sizeof(char) * 26);
    hipMemcpy(gpuAlphabet, cpuAlphabet, sizeof(char) * 26, hipMemcpyHostToDevice);

    char* gpuNumbers;
    hipMalloc((void**) &gpuNumbers, sizeof(char) * 10);
    hipMemcpy(gpuNumbers, cpuNumbers, sizeof(char) * 10, hipMemcpyHostToDevice);

    char* gpuEncryptedPass;
    hipMalloc((void**) &gpuEncryptedPass, sizeOfEncryptedPass);
    hipMemcpy(gpuEncryptedPass, encryptedPass, sizeOfEncryptedPass, hipMemcpyHostToDevice);

    char* gpuOutputPass;
    hipMalloc((void**) &gpuOutputPass, sizeOfEncryptedPass);

    // Launch CUDA kernel to decrypt the password
    decryptPass<<< dim3(26, 26, 1), dim3(10, 10, 1) >>>(gpuAlphabet, gpuNumbers, gpuEncryptedPass, gpuOutputPass);

    // Ensure the kernel has completed execution before proceeding
    hipDeviceSynchronize();
    printf("CUDA threads synchronization completed!\n");

    // Allocate and copy memory for the output password on CPU
    char* cpuOutputPass = (char*)malloc(sizeof(char) * 4);
    hipMemcpy(cpuOutputPass, gpuOutputPass, sizeOfEncryptedPass, hipMemcpyDeviceToHost);

    printf("---\n");
    printf("Results:\n");

    // Print the decrypted and encrypted passwords or an error message
    if (cpuOutputPass != NULL && cpuOutputPass[0] != '\0') {
        printf("1. Given Encrypted Pass: '%s'\n", encryptedPass);
        printf("2. Found Decrypted Pass: '%s'\n", cpuOutputPass);
    } else {
        printf("Unable to determine a password.\n");
    }

    // Free allocated GPU and CPU memory
    hipFree(gpuAlphabet);
    hipFree(gpuNumbers);
    hipFree(gpuEncryptedPass);
    hipFree(gpuOutputPass);
    free(cpuOutputPass);

    return 0;
}
