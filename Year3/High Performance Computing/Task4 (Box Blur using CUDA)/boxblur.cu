#include "hip/hip_runtime.h"
/*
Student Name: Nayan Raj Khanal
University ID: 2227486

The C code uses CUDA to accelerate a 3x3 box blur filter on a PNG image. 
It decodes the input image using the lodepng library, allocates GPU memory, and launches a CUDA kernel for parallelized image processing. 
After synchronizing threads, the blurred image is transferred back to the host and saved as a new PNG file. 
The code efficiently handles GPU memory and produces an output image with the applied blur effect.

1. Reading in an image file into a single or 2D array:
The lodepng library to decode a PNG image file named "input.png". 
The image data is stored in the image array, and the width and height of the image are obtained.

2. Allocating the correct amount of memory on the GPU based on input data. Memory is freed once used:
Device memory (d_input and d_output) is allocated using hipMalloc to store the image data on the GPU. 
The memory size is determined based on the image dimensions and the number of color channels (4 for RGBA). 
The hipFree function is used later to release this memory.

3. Applying Box filter on image in the kernel function:
The CUDA kernel function boxBlur operates on the GPU and applies a 3x3 box blur filter to each pixel in the image. 
It calculates the average color values of the 3x3 neighborhood for each pixel.

4. Return blurred image data from the GPU to the CPU:
After the GPU processing is complete, the blurred image data (d_output) is copied back from the GPU to the host (image) using hipMemcpyDeviceToHost.

5. Outputting the correct image with Box Blur applied as a file:
The blurred image data stored in the image array is encoded and saved as a new PNG file named "output.png" using the lodepng_encode32_file function. 
This file represents the original image with the box blur applied.

*/

// Header files
#include <stdio.h>
#include <stdlib.h>
#include "lodepng.h"

// CUDA kernel for applying a box blur filter to an image
__global__ void boxBlur(unsigned char *input, unsigned char *output, unsigned int width, unsigned int height)
{
    // Calculate the pixel coordinates in the output image based on block and thread indices
    int p_i = blockIdx.y * blockDim.y + threadIdx.y;
    int p_j = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the calculated coordinates are within the image dimensions
    if (p_i < height && p_j < width)
    {
        // Initialize variables to accumulate color channels and count neighboring pixels
        int r = 0, g = 0, b = 0;
        int count = 0;

        // Iterate over the 3x3 neighborhood centered at the current pixel
        for (int i = -1; i <= 1; ++i)
        {
            for (int j = -1; j <= 1; ++j)
            {
                // Calculate the coordinates of the neighboring pixel
                int neighbor_i = p_i + i;
                int neighbor_j = p_j + j;

                // Check if the neighboring pixel is within the image boundaries
                if (neighbor_i >= 0 && neighbor_i < height && neighbor_j >= 0 && neighbor_j < width)
                {
                    // Calculate the index of the neighboring pixel in the input image
                    int index = 4 * (neighbor_i * width + neighbor_j);
                    // Accumulate color channels and increment the count
                    r += input[index];
                    g += input[index + 1];
                    b += input[index + 2];
                    count++;
                }
            }
        }

        // Calculate the index of the current pixel in the output image
        int index = 4 * (p_i * width + p_j);
        // Compute the average color values for the 3x3 neighborhood and update the output image
        output[index] = r / count;
        output[index + 1] = g / count;
        output[index + 2] = b / count;
        // Copy the alpha channel from the input to the output (unchanged)
        output[index + 3] = input[index + 3];
    }
}


int main()
{
    unsigned char *image;       // Pointer to image data
    unsigned int width, height; // Width and height of the image

    // Decode PNG
    unsigned int error = lodepng_decode32_file(&image, &width, &height, "input.png");
    if (error)
    {
        fprintf(stderr, "Error %u: %s\n", error, lodepng_error_text(error));
        return 1;
    }

    // Allocate device memory for the image
    unsigned char *d_input, *d_output;
    hipMalloc((void **)&d_input, sizeof(unsigned char) * width * height * 4);
    hipMalloc((void **)&d_output, sizeof(unsigned char) * width * height * 4);

    // Copy image data from host to device
    hipMemcpy(d_input, image, sizeof(unsigned char) * width * height * 4, hipMemcpyHostToDevice);

    // Define block and grid dimensions
    dim3 blockSize(16, 16); // You can adjust the block size as needed
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    printf("Blurring image '%s'\n", "input.png");
    printf("---\n");

    // Launch the kernel on the GPU
    boxBlur<<<gridSize, blockSize>>>(d_input, d_output, width, height);

    // Ensure the kernel has completed execution before proceeding
    hipDeviceSynchronize();
    printf("CUDA threads synchronization completed!\n");

    // Copy the result back from device to host
    hipMemcpy(image, d_output, sizeof(unsigned char) * width * height * 4, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);

    // Encode and save the blurred image
    error = lodepng_encode32_file("output.png", image, width, height);
    if (error)
    {
        fprintf(stderr, "Error %u: %s\n", error, lodepng_error_text(error));
        return 1;
    }

    // Free the image data
    free(image);

    printf("Blurred image is saved as 'output.png'\n");

    return 0;
}
